#include <stdio.h>
#include <cstdlib>
#include <hipfft/hipfftXt.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>


// ********************************************************* //
// Error checking macros
// ********************************************************* //

#define ERR_CHECK(err_val) {                                    \
	hipError_t err = err_val;                                  \
	if (err != hipSuccess) {                                   \
		fprintf(stderr, "Error %s at line %d in file %s\n",	    \
				hipGetErrorString(err), __LINE__, __FILE__);   \
		exit(1);												\
	}                                                           \
}

#define FFT_ERR_CHECK(err_val) {                                \
	hipfftResult err = err_val;                                  \
    if (err != HIPFFT_SUCCESS) {                                 \
        fprintf(stderr, "Error %d at line %d in file %s\n",     \
                err, __LINE__, __FILE__);                       \
        exit(1);                                                \
    }                                                           \
}

// ********************************************************* //
// FT params for ...many plans
// ********************************************************* //

typedef struct FTParams{
        int numTranforms;
        int numLoops;
        int stride;
        int dist;
        int offset;
};

// ********************************************************* //
// Sample cuda kernels
// ********************************************************* //

__host__ __device__ double retVal(double val) {
    return val;
}

__global__ void copyVal(double *inData, double *outData) {
	outData[threadIdx.x] = inData[threadIdx.x];
}

// ********************************************************* //
// FT test functions
// ********************************************************* //

void fftIt2D(){
	hipDeviceReset();
	int dimSize = 6;
    int NX = dimSize*dimSize;

    int numTransform = std::round(sqrt(NX));
    int sqrtNX = std::round(sqrt(NX));
    int dims1D[] = {(NX)};
    int dims2D[] = {sqrtNX,sqrtNX};
	
	int rank = 1;

    int inembed[] = {NX}; 
    int onembed[] = {NX};
    int istride[] = {1,sqrtNX}; // Consecutive elements, same signal 
    int ostride[] = {1,sqrtNX}; //
    int idist[] = {sqrtNX,1}; // Consecutive signals
    int odist[] = {sqrtNX,1};

    hipfftHandle planMany, plan1D, plan2D;
    hipfftDoubleComplex *data_H1DFFT, *data_H2DF1DI, *data_HmanyFFT, *data_H0, *data_D;

    ERR_CHECK( hipMalloc( (hipfftDoubleComplex**) &data_D, sizeof(hipfftDoubleComplex)*NX) );
    data_H1DFFT = (hipfftDoubleComplex*) malloc(sizeof(hipfftDoubleComplex)*NX);
    data_H2DF1DI = (hipfftDoubleComplex*) malloc(sizeof(hipfftDoubleComplex)*NX);
    data_HmanyFFT = (hipfftDoubleComplex*) malloc(sizeof(hipfftDoubleComplex)*NX);
    data_H0 = (hipfftDoubleComplex*) malloc(sizeof(hipfftDoubleComplex)*NX);


    // ********************************************************* //
    // Create the input data
    // ********************************************************* //
    std::cout << "INPUT:\n";
    for(int ii=0; ii<sqrtNX; ++ii){
        for(int jj=0; jj<sqrtNX; ++jj){
            data_H0[jj + ii*sqrtNX].x = (double) ii;
            data_H0[jj + ii*sqrtNX].y = (double) jj;
            std::cout << data_H0[jj + ii*sqrtNX].x << " + 1i*" << data_H0[jj + ii*sqrtNX].y << "\t";
        }
        std::cout << "\n";
    }
    std::cout << "\n";

    ERR_CHECK( hipMemcpy(data_D, data_H0, sizeof(hipfftDoubleComplex) * NX, hipMemcpyHostToDevice));

    // ******************************************************************************** //
    // First, check the 1D FFT along the standard dimension
    // ******************************************************************************** //
    FFT_ERR_CHECK(hipfftPlan1d(&plan1D, dims2D[0], HIPFFT_Z2Z, numTransform));
    FFT_ERR_CHECK(hipfftExecZ2Z(plan1D, data_D, data_D, HIPFFT_FORWARD));
    ERR_CHECK(hipMemcpy(data_H1DFFT, data_D, sizeof(hipfftDoubleComplex) * NX, hipMemcpyDeviceToHost));
    std::cout << "OUTPUT 1D:\n";
    for(int ii=0; ii<sqrtNX; ++ii){
        for(int jj=0; jj<sqrtNX; ++jj){
            std::cout << data_H1DFFT[jj + ii*sqrtNX].x << " + 1i*" << data_H1DFFT[jj + ii*sqrtNX].y << "\t";
        }
        std::cout << "\n";
    }

/*
    //Check the inverse FFT for errors
    FFT_ERR_CHECK(cufftExecZ2Z(plan1D, data_D, data_D, CUFFT_INVERSE));
    ERR_CHECK(cudaMemcpy(data_H1DFFT, data_D, sizeof(cufftDoubleComplex) * NX, cudaMemcpyDeviceToHost));

    std::cout << "OUTPUT 1D Inverse:\n";
    for(int ii=0; ii<sqrtNX; ++ii){
        for(int jj=0; jj<sqrtNX; ++jj){
            std::cout << data_H[jj + ii*sqrtNX].x/sqrtNX << " + 1i*" << data_H[jj + ii*sqrtNX].y/sqrtNX << "\t";
        }
        std::cout << "\n";
    }
*/
    //Overwrite GPU data to original values
    ERR_CHECK( hipMemcpy(data_D, data_H0, sizeof(hipfftDoubleComplex) * NX, hipMemcpyHostToDevice));

    // ******************************************************************************** //
    // ******************************************************************************** //
    // Next, check the Many FFT along the same expected dimension
    // ******************************************************************************** //

    FFT_ERR_CHECK(hipfftPlanMany(&planMany, rank, dims2D, inembed, istride[0], idist[0], onembed, ostride[0], odist[0], HIPFFT_Z2Z, numTransform));
    FFT_ERR_CHECK(hipfftExecZ2Z(planMany, data_D, data_D, HIPFFT_FORWARD));

    //ERR_CHECK(cudaDeviceSynchronize());
    ERR_CHECK(hipMemcpy(data_HmanyFFT, data_D, sizeof(hipfftDoubleComplex) * NX, hipMemcpyDeviceToHost));

    std::cout << "OUTPUT MANY 1D:\n";
    for(int ii=0; ii<sqrtNX; ++ii){
        for(int jj=0; jj<sqrtNX; ++jj){
            std::cout << data_HmanyFFT[jj + ii*sqrtNX].x << " + 1i*" << data_HmanyFFT[jj + ii*sqrtNX].y << "\t";
        }
        std::cout << "\n";
    }

	try {
		for (int ii=0; ii<NX; ++ii){
			assert( (data_H1DFFT[ii].x - data_HmanyFFT[ii].x) < 1e-7  );
			assert( (data_H1DFFT[ii].y - data_HmanyFFT[ii].y) < 1e-7  );
		}
	} catch (const char* msg) {
		std::cerr << msg << std::endl;
	}
    //Overwrite GPU data to original values
    ERR_CHECK( hipMemcpy(data_D, data_H0, sizeof(hipfftDoubleComplex) * NX, hipMemcpyHostToDevice));

    // ******************************************************************************** //
    // Check the 2D FFT Forward, 1D FFT back
    // ******************************************************************************** //
    FFT_ERR_CHECK(hipfftPlan2d(&plan2D, dims2D[0], dims2D[1], HIPFFT_Z2Z));
    FFT_ERR_CHECK(hipfftExecZ2Z(plan2D, data_D, data_D, HIPFFT_FORWARD));
    FFT_ERR_CHECK(hipfftExecZ2Z(plan1D, data_D, data_D, HIPFFT_BACKWARD));

    ERR_CHECK(hipMemcpy(data_H2DF1DI, data_D, sizeof(hipfftDoubleComplex) * NX, hipMemcpyDeviceToHost));
    std::cout << "OUTPUT 2DF-1DI:\n";
    for(int ii=0; ii<sqrtNX; ++ii){
        for(int jj=0; jj<sqrtNX; ++jj){
            std::cout << data_H2DF1DI[jj + ii*sqrtNX].x/sqrtNX << " + 1i*" << data_H2DF1DI[jj + ii*sqrtNX].y/sqrtNX << "\t";
        }
        std::cout << "\n";
    }
    //Overwrite GPU data to original values
    ERR_CHECK( hipMemcpy(data_D, data_H0, sizeof(hipfftDoubleComplex) * NX, hipMemcpyHostToDevice));

    // ******************************************************************************** //
    // ******************************************************************************** //
    // Lastly, check the Many FFT along the other dimension
    // ******************************************************************************** //

    FFT_ERR_CHECK(hipfftPlanMany(&planMany, rank, dims2D, inembed, istride[1], idist[1], onembed, ostride[1], odist[1], HIPFFT_Z2Z, sqrtNX));
/*	for(int ii=0; ii<sqrtNX; ++ii){
    	FFT_ERR_CHECK(cufftExecZ2Z(planMany, &data_D[0], &data_D[0], CUFFT_FORWARD));
	}*/
    FFT_ERR_CHECK(hipfftExecZ2Z(planMany, data_D, data_D, HIPFFT_FORWARD));

    //ERR_CHECK(cudaDeviceSynchronize());
    ERR_CHECK(hipMemcpy(data_HmanyFFT, data_D, sizeof(hipfftDoubleComplex) * NX, hipMemcpyDeviceToHost));

    std::cout << "OUTPUT MANY 1D Other:\n";
    for(int ii=0; ii<sqrtNX; ++ii){
        for(int jj=0; jj<sqrtNX; ++jj){
            std::cout << data_HmanyFFT[jj + ii*sqrtNX].x << " + 1i*" << data_HmanyFFT[jj + ii*sqrtNX].y << "\t";
        }
        std::cout << "\n";
    }
	try {
		for (int ii=0; ii<NX; ++ii){
			//std::cout << ( (data_H2DF1DI[ii].x/sqrtNX - data_HmanyFFT[ii].x) < 1e-7  ) << "\n";
		    assert( (data_H2DF1DI[ii].x/sqrtNX - data_HmanyFFT[ii].x)  < 1e-7 );
			assert( (data_H2DF1DI[ii].y/sqrtNX - data_HmanyFFT[ii].y)  < 1e-7 );
		}
	} catch (const char* msg) {
		std::cerr << msg << std::endl;
	}

    //Overwrite GPU data to original values
    ERR_CHECK( hipMemcpy(data_D, data_H0, sizeof(hipfftDoubleComplex) * NX, hipMemcpyHostToDevice));

    // ******************************************************************************** //
    // ******************************************************************************** //
    // Free stuff
    // ******************************************************************************** //

    hipfftDestroy(planMany);hipfftDestroy(plan1D);hipfftDestroy(plan2D);
    hipFree(data_D);
    free(data_HmanyFFT);free(data_H0);
    free(data_H2DF1DI);free(data_H1DFFT);
}


void fftIt3D(){
    int dimLength = 5;
	int NX = dimLength*dimLength*dimLength;

    int cbrtNX = std::cbrt(NX);
    int numTransform = cbrtNX*cbrtNX;

    int paramsMatrix[3][5] = {{cbrtNX*cbrtNX,1,1,cbrtNX,0},{cbrtNX,cbrtNX,cbrtNX,1,cbrtNX*cbrtNX},{cbrtNX*cbrtNX,1,cbrtNX*cbrtNX,1,0}};
    FTParams params[3];

    for(int ii=0; ii<3; ++ii){
        params[ii].numTranforms = paramsMatrix[ii][0];
        params[ii].numLoops = paramsMatrix[ii][1];
        params[ii].stride = paramsMatrix[ii][2];
        params[ii].dist = paramsMatrix[ii][3];
        params[ii].offset = paramsMatrix[ii][4];
    }

    int dims[] = {NX};
    int dims3D[] = {cbrtNX,cbrtNX,cbrtNX};

    int inembed[] = {cbrtNX,cbrtNX,cbrtNX};
    int onembed[] = {cbrtNX,cbrtNX,cbrtNX};
    int istride[] = {1,cbrtNX,cbrtNX*cbrtNX}; // Indexed value is respective dimensionality of the transform along a specific dimension.
    int ostride[] = {1,cbrtNX,cbrtNX*cbrtNX};
    int idist[] = {cbrtNX,1,1}; // [Here][] // The next dimension
    int odist[] = {cbrtNX,1,1};

    hipfftHandle planMany, plan1D, plan3D;
    hipfftDoubleComplex *data_H1DFFT, *data_HmanyFFT, *data_H0, *data_D;

    ERR_CHECK( hipMalloc( (hipfftDoubleComplex**) &data_D, sizeof(hipfftDoubleComplex)*NX) );

    data_H1DFFT = (hipfftDoubleComplex*) malloc(sizeof(hipfftDoubleComplex)*NX);
    data_HmanyFFT = (hipfftDoubleComplex*) malloc(sizeof(hipfftDoubleComplex)*NX);
    data_H0 = (hipfftDoubleComplex*) malloc(sizeof(hipfftDoubleComplex)*NX);


    // ******************************************************************************** //
    // Create the input data
    // ******************************************************************************** //
    std::cout << "INPUT:\n";
    for(int ii=0; ii<cbrtNX; ++ii){
		std::cout << "C(:,:," << ii+1 << ")=[";
        for(int jj=0; jj<cbrtNX; ++jj){
            for(int kk=0; kk<cbrtNX; ++kk){
                data_H0[kk + cbrtNX*(jj + ii*cbrtNX)].x = (double) ii;
                data_H0[kk + cbrtNX*(jj + ii*cbrtNX)].y = (double) jj;//(double) jj;
                std::cout << data_H0[kk + cbrtNX*(jj + ii*cbrtNX)].x << " + 1i*" << data_H0[kk + cbrtNX*(jj + ii*cbrtNX)].y << "\t";
            }
            std::cout << "\n";
        }
        std::cout << "]\n";
    }
    std::cout << "\n --- \n";

    ERR_CHECK( hipMemcpy(data_D, data_H0, sizeof(hipfftDoubleComplex) * NX, hipMemcpyHostToDevice));

    // ******************************************************************************** //
    // First, check the 1D FFT along the standard dimension
    // ******************************************************************************** //
    FFT_ERR_CHECK(hipfftPlan1d(&plan1D, cbrtNX, HIPFFT_Z2Z, cbrtNX*cbrtNX));
    FFT_ERR_CHECK(hipfftExecZ2Z(plan1D, data_D, data_D, HIPFFT_FORWARD));
    ERR_CHECK(hipMemcpy(data_H1DFFT, data_D, sizeof(hipfftDoubleComplex) * NX, hipMemcpyDeviceToHost));
    std::cout << "OUTPUT 1D_1:\n";
    for(int ii=0; ii<cbrtNX; ++ii){
        for(int jj=0; jj<cbrtNX; ++jj){
            for(int kk=0; kk<cbrtNX; ++kk){
                std::cout << data_H1DFFT[kk + cbrtNX*(jj + ii*cbrtNX)].x << " + 1i*" << data_H1DFFT[kk + cbrtNX*(jj + ii*cbrtNX)].y << "\t";
            }
            std::cout << "\n";
        }
        std::cout << "\n";
    }
    std::cout << "\n --- \n";

    //Overwrite GPU data to original values
    ERR_CHECK( hipMemcpy(data_D, data_H0, sizeof(hipfftDoubleComplex) * NX, hipMemcpyHostToDevice));

    // ******************************************************************************** //
    // ******************************************************************************** //
    // Next, check the Many FFT along the same expected dimension
    // ******************************************************************************** //
    int tDim = 0; //Transform dimension
	
	int dims2D[] = {cbrtNX,cbrtNX};
   	FFT_ERR_CHECK(hipfftPlanMany(&planMany, 1, dims2D, inembed, cbrtNX, 1, onembed, cbrtNX, 1, HIPFFT_Z2Z, cbrtNX));
    for (int ii=0; ii<cbrtNX; ++ii){
		FFT_ERR_CHECK(hipfftExecZ2Z(planMany, &data_D[ii*cbrtNX*cbrtNX], &data_D[ii*cbrtNX*cbrtNX] , HIPFFT_FORWARD));
	}

    /*for (int ft=0; ft < params[tDim].numLoops; ++ft){
        FFT_ERR_CHECK(cufftPlanMany(&planMany, 1, dims, inembed, params[tDim].stride, params[tDim].dist, onembed, params[tDim].stride, params[tDim].dist, CUFFT_Z2Z, params[tDim].numTranforms));
        //FFT_ERR_CHECK(cufftExecZ2Z(planMany, data_D + ((int) pow(cbrtNX,tDim)), data_D + ((int) pow(cbrtNX,tDim)), CUFFT_FORWARD));
        FFT_ERR_CHECK(cufftExecZ2Z(planMany, &data_D[ft*params[tDim].offset], &data_D[ft*params[tDim].offset], CUFFT_FORWARD));
    }*/
    //ERR_CHECK(cudaDeviceSynchronize());
    ERR_CHECK(hipMemcpy(data_HmanyFFT, data_D, sizeof(hipfftDoubleComplex) * NX, hipMemcpyDeviceToHost));

    std::cout << "OUTPUT MANY 1D:\n";
    for(int ii=0; ii<cbrtNX; ++ii){
            for(int jj=0; jj<cbrtNX; ++jj){
                for(int kk=0; kk<cbrtNX; ++kk){
                    std::cout << data_HmanyFFT[kk + cbrtNX*(jj + ii*cbrtNX)].x << " + 1i*" << data_HmanyFFT[kk + cbrtNX*(jj + ii*cbrtNX)].y << "\t";
                }
                std::cout << "\n";
            }
            std::cout << "\n";
        }
        std::cout << "\n --- \n";

    //Overwrite GPU data to original values
    ERR_CHECK( hipMemcpy(data_D, data_H0, sizeof(hipfftDoubleComplex) * NX, hipMemcpyHostToDevice));

    // ******************************************************************************** //
    // ******************************************************************************** //
    // Free stuff
    // ******************************************************************************** //

    hipfftDestroy(planMany);hipfftDestroy(plan1D);hipfftDestroy(plan3D);
    hipFree(data_D);
    free(data_HmanyFFT);free(data_H1DFFT);free(data_H0);
}

void fftMulti3D(){
   	int GPU_N;
    hipGetDeviceCount(&GPU_N);
	
	std::cout << "Num. Devices = " << GPU_N << "\n";

	hipfftHandle plan_input; hipfftResult result;
	FFT_ERR_CHECK( hipfftCreate(&plan_input) );

	int nGPUs = 2;
	int* whichGPUs = (int*) malloc(sizeof(int)*nGPUs);
	for (int i=0; i<nGPUs; ++i){
		whichGPUs[i] = i;
	}
	FFT_ERR_CHECK( hipfftXtSetGPUs(plan_input, nGPUs, whichGPUs) );
    

    //Print the device information to run the code
     for (int i = 0 ; i < nGPUs ; i++)
     {
         hipDeviceProp_t deviceProp;
         ERR_CHECK( hipGetDeviceProperties(&deviceProp, whichGPUs[i]) );
         printf("GPU Device %d: \"%s\" with compute capability %d.%d\n", whichGPUs[i], deviceProp.name, deviceProp.major, deviceProp.minor);

     }

	size_t* worksize = (size_t*) malloc(sizeof(size_t)*nGPUs);
    hipfftDoubleComplex *host_data_input, *host_data_output;

    int nx=4,ny,nz;
	ny = nx; nz = ny;

    int size_of_data = sizeof(hipfftDoubleComplex) * nx * ny * nz;
    host_data_input = (hipfftDoubleComplex*) malloc(size_of_data);
    host_data_output = (hipfftDoubleComplex*) malloc(size_of_data);

	for(int ii=0; ii<nx; ++ii){	
		std::cout << "C(:,:," << ii+1 << ")=[";
		for(int jj=0; jj<ny; ++jj){	
			for(int kk=0; kk<nz; ++kk){
				host_data_input[kk + ny*(jj + nx*ii)].x = kk;//(double) kk + ny*(jj + nx*ii);
				host_data_input[kk + ny*(jj + nx*ii)].y = jj;//(double) kk + ny*(jj + nx*ii);
 				std::cout << host_data_input[kk + ny*(jj + nx*ii)].x << "+1i*" << host_data_input[kk + ny*(jj + nx*ii)].y << "  ";
			}	std::cout << "\n";
		}	std::cout << "]\n";
	}

    //FFT_ERR_CHECK( cufftMakePlan3d (plan_input, nz, ny, nx, CUFFT_Z2Z, worksize) );
	int rank = 3; //3D
	long long int dims[3] = {nx,ny,nz};
    FFT_ERR_CHECK( 
		hipfftXtMakePlanMany(plan_input, rank, dims, NULL, 0LL, 0LL, HIP_C_64F, NULL, 0LL, 0LL, HIP_C_64F, 1LL, worksize, HIP_C_64F) 
	);//not supported under cufft7.5
    
	hipLibXtDesc *device_data_input;
    FFT_ERR_CHECK( hipfftXtMalloc (plan_input, &device_data_input, HIPFFT_XT_FORMAT_INPLACE) );
	FFT_ERR_CHECK( hipfftXtMemcpy (plan_input, device_data_input, host_data_input, HIPFFT_COPY_HOST_TO_DEVICE) );
	FFT_ERR_CHECK( hipfftXtExecDescriptorZ2Z (plan_input, device_data_input, device_data_input, HIPFFT_FORWARD) );
	FFT_ERR_CHECK( hipfftXtMemcpy (plan_input, host_data_output, device_data_input, HIPFFT_COPY_DEVICE_TO_HOST) );
	
	for(int ii=0; ii<nx; ++ii){			
		std::cout << "C(:,:," << ii+1 << ")=[";
		for(int jj=0; jj<ny; ++jj){	
			for(int kk=0; kk<nz; ++kk){
 				std::cout << host_data_output[kk + ny*(jj + nx*ii)].x << "+1i*" << host_data_output[kk + ny*(jj + nx*ii)].y << "  ";
			}	std::cout << "\n";
		}	std::cout << "]\n";
	}

    result = hipfftXtFree(device_data_input);
	result = hipfftDestroy(plan_input);

	free(host_data_input); free(host_data_output);
}

int main(void) {

    fftMulti3D();
    ERR_CHECK(hipDeviceReset());

    return (0);
}
